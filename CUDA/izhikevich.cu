#include "hip/hip_runtime.h"
/*=======================================
  Since : May/19/2008
  Update: <2016/02/26>

float
  =======================================*/
#include "izhikevich.h"


__device__ double dv(double v,double u, double inp){
  return 0.04*v*v +5*v +140 -u +inp;
}

__device__ double du(double v,double u){
  return a*(b*v-u);
}



__global__ void init(double *v,double *u,int *spike, double *inp,int *spikecnt)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  v[i] = V0;
  u[i] = U0;
  inp[i] = 0;
  spike[i] = 0;
  spikecnt[i]=0;

}

__global__ void calv(double *v,double *u,int *spike, double *inp, double *t,int *spikecnt,double TH,int NUM,int sigma)
{



  int i = threadIdx.x + blockIdx.x * blockDim.x;
  inp[i] = I0;

  double kv1 = DT*dv(v[i],u[i], inp[i]);
  double ku1 = DT*du(v[i],u[i]);

  double kv2 = DT*dv(v[i]+kv1*0.5,u[i]+ku1*0.5, inp[i]);
  double ku2 = DT*du(v[i]+kv1*0.5,u[i]+ku1*0.5);

  double kv3 = DT*dv(v[i]+kv2*0.5,u[i]+ku2*0.5, inp[i]);
  double ku3 = DT*du(v[i]+kv2*0.5,u[i]+ku2*0.5);

  double kv4 = DT*dv(v[i]+kv3, u[i]+ku3, inp[i]);
  double ku4 = DT*du(v[i]+kv3, u[i]+ku3);

  v[i] += (kv1 + 2.0*kv2 + 2.0*kv3 + kv4)/6.0;
  u[i] += (ku1 + 2.0*ku2 + 2.0*ku3 + ku4)/6.0;


  if(v[i] > TH){
    v[i] =c ;
    u[i] = u[i]+d;
    spike[i] = spike[i]+1;
    spikecnt[i]=spike[i];

  }
  if(int(t[0])%10==0){
    spike[i]=0;
  }
}




void Simulation::sim()
{
    int count = 0;

    int size_d = sizeof(double)*NUM;

    double *v,*d_v;
    double *u,*d_u;
    double *inp,*d_inp;
    v = (double *)malloc(size_d);
    u = (double *)malloc(size_d);
    inp = (double *)malloc(size_d);
    hipMalloc((void **)&d_v, size_d);
    hipMalloc((void **)&d_u, size_d);
    hipMalloc((void **)&d_inp, size_d);




    double t = 0.0;
    double *d_t;
    hipMalloc((void **)&d_t, sizeof(double));

    int *spike,*d_spike;
    int *spikecnt,*d_spikecnt;

    //    FILE *fp1;
    //fp1=fopen("cuda_double_izh_volt.txt","w");


    int size_i = sizeof(int)*NUM;
    spike  = (int *)malloc(size_i);
    hipMalloc((void **)&d_spike, size_i);
    spikecnt  = (int *)malloc(size_i);
    hipMalloc((void **)&d_spikecnt, size_i);



    init<<<NUM/Threads,Threads>>>(d_v,d_u, d_spike, d_inp,d_spikecnt);

    for(;;){

      calv<<<NUM/Threads,Threads>>>(d_v,d_u, d_spike, d_inp, d_t, d_spikecnt,TH,NUM,sigma);

      hipMemcpy(v, d_v, size_d, hipMemcpyDeviceToHost);
      hipMemcpy(u, d_u, size_d, hipMemcpyDeviceToHost);
      hipMemcpy(inp, d_inp, size_d, hipMemcpyDeviceToHost);
      hipMemcpy(spike, d_spike, size_i, hipMemcpyDeviceToHost);
      hipMemcpy(spikecnt, d_spikecnt, size_i, hipMemcpyDeviceToHost);

      fprintf(fp1,"%lf \n",v[0]); 


      count++;
      t = count * DT;
      if( t > TEND){
	break;
      }
    }
    free(v);
    free(u);
    free(spike);
    free(spikecnt);
    hipFree(d_v);
    hipFree(d_u);
    hipFree(d_spike);
    hipFree(d_spikecnt);

}



int main(int argc, char* argv[]){
 Simulation sim;
 sim.sim();
 return(0);
 }
